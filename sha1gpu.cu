#include "hip/hip_runtime.h"
#include"hip/hip_runtime.h"
#include "Common1.h"
#include""
#include"stdio.h"
#include"time.h"
typedef struct {
	unsigned long state[5];
} sha1_gpu_context;
typedef struct {
	float malloctime;
	float copytime;
	float kerneltime;
}time_cuda_collect;
time_cuda_collect A1 = { 0,0,0};
#define S(x,n) ((x << n) | ((x & 0xFFFFFFFF) >> (32 - n)))
#define R(t) \
	temp = extended[block_index + t -  3] ^ extended[block_index + t - 8] ^     \
		   extended[block_index + t - 14] ^ extended[block_index + t - 16]; \
	extended[block_index + t] = S(temp,1); \
//Another methods
__device__ inline unsigned long f1(unsigned long x, unsigned long y, unsigned long z) { return(z ^ (x&(y^z))); }// (x & y ) | ( ~x & z)
__device__ inline unsigned long f2(unsigned long x, unsigned long y, unsigned long z) { return(x^y^z); }
__device__ inline unsigned long f3(unsigned long x, unsigned long y, unsigned long z) { return((x&y)|(z&(x|y)));}
__device__ inline unsigned long f4(unsigned long x, unsigned long y, unsigned long z) { return(x^y^z); }
//
__constant__ unsigned long C1 = 0x5A827999;
__constant__ unsigned long C2 = 0x6Ed9EBA1;
__constant__ unsigned long C3 = 0x8F1BBCDC;
__constant__ unsigned long C4 = 0xCA62C1D6;
__device__ unsigned long SST(unsigned long x, int n) { return((x << n)|((x & 0xFFFFFFFF)>>(32 - n))); }
__device__ unsigned long p1(unsigned long a, unsigned long b, unsigned long c, unsigned long d, unsigned long x){ return(SST(a,5)+f1(b,c,d)+C1+x);  }
__device__ unsigned long p2(unsigned long a, unsigned long b, unsigned long c, unsigned long d, unsigned long x){ return(SST(a,5)+f2(b,c,d)+C2+x);   }
__device__ unsigned long p3(unsigned long a, unsigned long b, unsigned long c, unsigned long d, unsigned long x){ return(SST(a, 5) + f3(b, c, d) + C3 + x); }
__device__ unsigned long p4(unsigned long a, unsigned long b, unsigned long c, unsigned long d, unsigned long x){ return(SST(a, 5) + f4(b, c, d) + C4 + x); }
__device__ void sha1_gpu_process2(sha1_gpu_context *ctx, unsigned long W[80]) {
	unsigned long A, B, C, D, E;
	A = ctx->state[0];
	B = ctx->state[1];
	C = ctx->state[2];
	D = ctx->state[3];
	E = ctx->state[4];

	for (int t = 0; t < 16; t++) {
		if (5*t < 20) {
			E = E + p1(A, B, C, D, W[0+5*t]); B = SST(B, 30);
			D = D + p1(E, A, B, C, W[1+5*t]); A = SST(A, 30);
			C = C + p1(D, E, A, B, W[2+5*t]); E = SST(E, 30);
			B = B + p1(C, D, E, A, W[3+5*t]); D = SST(D, 30);
			A = A + p1(B, C, D, E, W[4+5*t]); C = SST(C, 30);
		}
		if ((5*t < 40)&&(5*t>=20)) {
			E = E + p2(A, B, C, D, W[0 + 5 * t]); B = SST(B, 30);
			D = D + p2(E, A, B, C, W[1 + 5 * t]); A = SST(A, 30);
			C = C + p2(D, E, A, B, W[2 + 5 * t]); E = SST(E, 30);
			B = B + p2(C, D, E, A, W[3 + 5 * t]); D = SST(D, 30);
			A = A + p2(B, C, D, E, W[4 + 5 * t]); C = SST(C, 30);
		}
		if ((5*t < 60)&&(5*t>=40)) {
			E = E + p3(A, B, C, D, W[0 + 5 * t]); B = SST(B, 30);
			D = D + p3(E, A, B, C, W[1 + 5 * t]); A = SST(A, 30);
			C = C + p3(D, E, A, B, W[2 + 5 * t]); E = SST(E, 30);
			B = B + p3(C, D, E, A, W[3 + 5 * t]); D = SST(D, 30);
			A = A + p3(B, C, D, E, W[4 + 5 * t]); C = SST(C, 30);
		}
		if ((5*t < 80)&&(5*t>=60)) {
			E = E + p4(A, B, C, D, W[0 + 5 * t]); B = SST(B, 30);
			D = D + p4(E, A, B, C, W[1 + 5 * t]); A = SST(A, 30);
			C = C + p4(D, E, A, B, W[2 + 5 * t]); E = SST(E, 30);
			B = B + p4(C, D, E, A, W[3 + 5 * t]); D = SST(D, 30);
			A = A + p4(B, C, D, E, W[4 + 5 * t]); C = SST(C, 30);
		}
	}
	//printf("%x,%x,%x,%x,%x\n", ctx->state[0], ctx->state[1], ctx->state[2], ctx->state[3], ctx->state[4]);
	ctx->state[0] += A;
	ctx->state[1] += B;
	ctx->state[2] += C;
	ctx->state[3] += D;
	ctx->state[4] += E;
}
__device__ void sha1_gpu_process(sha1_gpu_context *ctx, unsigned long W[80])
{

	__shared__ unsigned long A, B, C, D, E;
	A = ctx->state[0];
	B = ctx->state[1];
	C = ctx->state[2];
	D = ctx->state[3];
	E = ctx->state[4];
	// 4 rounds calculation defination
#define P(a,b,c,d,e,x)                                  \
{                                                       \
    e += S(a,5) + F(b,c,d) + K + x; b = S(b,30);        \
}

	//0~19 rounds corresponding function and data
#define F(x,y,z) (z ^ (x & (y ^ z)))
#define K 0x5A827999

	P(A, B, C, D, E, W[0]);
	P(E, A, B, C, D, W[1]);
	P(D, E, A, B, C, W[2]);
	P(C, D, E, A, B, W[3]);
	P(B, C, D, E, A, W[4]);
	P(A, B, C, D, E, W[5]);
	P(E, A, B, C, D, W[6]);
	P(D, E, A, B, C, W[7]);
	P(C, D, E, A, B, W[8]);
	P(B, C, D, E, A, W[9]);
	P(A, B, C, D, E, W[10]);
	P(E, A, B, C, D, W[11]);
	P(D, E, A, B, C, W[12]);
	P(C, D, E, A, B, W[13]);
	P(B, C, D, E, A, W[14]);
	P(A, B, C, D, E, W[15]);
	P(E, A, B, C, D, W[16]);
	P(D, E, A, B, C, W[17]);
	P(C, D, E, A, B, W[18]);
	P(B, C, D, E, A, W[19]);

#undef K
#undef F
	//20~39 rounds corresponding function and data
#define F(x,y,z) (x ^ y ^ z)
#define K 0x6ED9EBA1

	P(A, B, C, D, E, W[20]);
	P(E, A, B, C, D, W[21]);
	P(D, E, A, B, C, W[22]);
	P(C, D, E, A, B, W[23]);
	P(B, C, D, E, A, W[24]);
	P(A, B, C, D, E, W[25]);
	P(E, A, B, C, D, W[26]);
	P(D, E, A, B, C, W[27]);
	P(C, D, E, A, B, W[28]);
	P(B, C, D, E, A, W[29]);
	P(A, B, C, D, E, W[30]);
	P(E, A, B, C, D, W[31]);
	P(D, E, A, B, C, W[32]);
	P(C, D, E, A, B, W[33]);
	P(B, C, D, E, A, W[34]);
	P(A, B, C, D, E, W[35]);
	P(E, A, B, C, D, W[36]);
	P(D, E, A, B, C, W[37]);
	P(C, D, E, A, B, W[38]);
	P(B, C, D, E, A, W[39]);

#undef K
#undef F
	//40~59 rounds corresponding function and data
#define F(x,y,z) ((x & y) | (z & (x | y)))
#define K 0x8F1BBCDC

	P(A, B, C, D, E, W[40]);
	P(E, A, B, C, D, W[41]);
	P(D, E, A, B, C, W[42]);
	P(C, D, E, A, B, W[43]);
	P(B, C, D, E, A, W[44]);
	P(A, B, C, D, E, W[45]);
	P(E, A, B, C, D, W[46]);
	P(D, E, A, B, C, W[47]);
	P(C, D, E, A, B, W[48]);
	P(B, C, D, E, A, W[49]);
	P(A, B, C, D, E, W[50]);
	P(E, A, B, C, D, W[51]);
	P(D, E, A, B, C, W[52]);
	P(C, D, E, A, B, W[53]);
	P(B, C, D, E, A, W[54]);
	P(A, B, C, D, E, W[55]);
	P(E, A, B, C, D, W[56]);
	P(D, E, A, B, C, W[57]);
	P(C, D, E, A, B, W[58]);
	P(B, C, D, E, A, W[59]);

#undef K
#undef F
	//60~79 rounds function 
#define F(x,y,z) (x ^ y ^ z)
#define K 0xCA62C1D6

	P(A, B, C, D, E, W[60]);
	P(E, A, B, C, D, W[61]);
	P(D, E, A, B, C, W[62]);
	P(C, D, E, A, B, W[63]);
	P(B, C, D, E, A, W[64]);
	P(A, B, C, D, E, W[65]);
	P(E, A, B, C, D, W[66]);
	P(D, E, A, B, C, W[67]);
	P(C, D, E, A, B, W[68]);
	P(B, C, D, E, A, W[69]);
	P(A, B, C, D, E, W[70]);
	P(E, A, B, C, D, W[71]);
	P(D, E, A, B, C, W[72]);
	P(C, D, E, A, B, W[73]);
	P(B, C, D, E, A, W[74]);
	P(A, B, C, D, E, W[75]);
	P(E, A, B, C, D, W[76]);
	P(D, E, A, B, C, W[77]);
	P(C, D, E, A, B, W[78]);
	P(B, C, D, E, A, W[79]);

#undef K
#undef F
	// Final operation:Add this chunk's hash to result so far
	ctx->state[0] += A;
	ctx->state[1] += B;
	ctx->state[2] += C;
	ctx->state[3] += D;
	ctx->state[4] += E;

}


/*
* Process extended block in GPU,analysis: there are no existing parallel methods for 
* the inputs are closely related to the output
*/


void __global__  sha1_kernel_global(unsigned char *data, sha1_gpu_context *ctx, int total_threads, unsigned long *extended)
{
	int thread_index = threadIdx.x + blockDim.x * blockIdx.x;
	int e_index = thread_index * 80;
	int block_index = thread_index * 64;//512 byte is a block
	unsigned long temp, t;
	if (thread_index > total_threads - 1)
		return;

	/*
	* load 32 to 80 blocks
	*/
	GET_UINT32_BE(extended[e_index], data + block_index, 0);
	GET_UINT32_BE(extended[e_index + 1], data + block_index, 4);
	GET_UINT32_BE(extended[e_index + 2], data + block_index, 8);
	GET_UINT32_BE(extended[e_index + 3], data + block_index, 12);
	GET_UINT32_BE(extended[e_index + 4], data + block_index, 16);
	GET_UINT32_BE(extended[e_index + 5], data + block_index, 20);
	GET_UINT32_BE(extended[e_index + 6], data + block_index, 24);
	GET_UINT32_BE(extended[e_index + 7], data + block_index, 28);
	GET_UINT32_BE(extended[e_index + 8], data + block_index, 32);
	GET_UINT32_BE(extended[e_index + 9], data + block_index, 36);
	GET_UINT32_BE(extended[e_index + 10], data + block_index, 40);
	GET_UINT32_BE(extended[e_index + 11], data + block_index, 44);
	GET_UINT32_BE(extended[e_index + 12], data + block_index, 48);
	GET_UINT32_BE(extended[e_index + 13], data + block_index, 52);
	GET_UINT32_BE(extended[e_index + 14], data + block_index, 56);
	GET_UINT32_BE(extended[e_index + 15], data + block_index, 60);

	for (t = 16; t < 80; t++) {
		temp = extended[e_index + t - 3] ^ extended[e_index + t - 8] ^
			extended[e_index + t - 14] ^ extended[e_index + t - 16];
		extended[e_index + t] = S(temp, 1);
	}
	/* Wait for the last thread and compute intermediate hash values of extended blocks */
	__syncthreads();
	if (thread_index == total_threads - 1) {
		for (t = 0; t < total_threads; t++)
			sha1_gpu_process(ctx, (unsigned long*)&extended[t * 80]);

	}
}


void sha1_gpu_global1(unsigned char *input, unsigned long size, unsigned char *output, int proc)
{
	int total_threads;
	int blocks_per_grid;
	int threads_per_block;
	int pad, size_be;
	int total_datablocks;
	int i, k;
	unsigned char *d_message;
	unsigned long *d_extended;
	sha1_gpu_context ctx, *d_ctx;
	//Initialize the parameter
	ctx.state[0] = 0x67452301;
	ctx.state[1] = 0xEFCDAB89;
	ctx.state[2] = 0x98BADCFE;
	ctx.state[3] = 0x10325476;
	ctx.state[4] = 0xC3D2E1F0;

	pad = padding_256(size);//To pad depended on size
	threads_per_block = proc;
	blocks_per_grid = 1;
	total_datablocks = (size + pad + 8) / 64; //64;
	//printf("total_datablocks is %d\n", total_datablocks);
	//Limit the number of total_threads
	if (total_datablocks > threads_per_block)
		total_threads = threads_per_block;//In this program is 1
	else
		total_threads = total_datablocks;
	//printf("total_threads is %d\n", total_threads);
	size_be = LETOBE32(size * 8);
	/* allocate enough memory*/
	clock_t start1, finish1;
	start1 = clock();
	hipMalloc((void**)&d_extended, proc * 80 * sizeof(unsigned long));
	hipMalloc((void**)&d_message, size + pad + 8);
	hipMalloc((void**)&d_ctx, sizeof(sha1_gpu_context));
	finish1 = clock();
	A1.malloctime = (finish1 - start1) / CLOCKS_PER_SEC;
	clock_t start2, finish2;
	start2 = clock();
	hipMemcpy(d_ctx, &ctx, sizeof(sha1_gpu_context), hipMemcpyHostToDevice);
	hipMemcpy(d_message, input, size, hipMemcpyHostToDevice);
	hipMemset(d_message + size, 0x80, 1);
	hipMemset(d_message + size + 1, 0, pad + 7);
	hipMemcpy(d_message + size + pad + 4, &size_be, 4, hipMemcpyHostToDevice);
	finish2 = clock();
	A1.copytime = (finish2 - start2) / CLOCKS_PER_SEC;
	/*
	* run the algorithm
	*/
	i = 0;
    k = total_datablocks / total_threads;
	clock_t start3, finish3;
	start3 = clock();
	if (k - 1 > 0) {
		for (i = 0; i < k; i++) {
			sha1_kernel_global << <total_datablocks, proc >> >(d_message + threads_per_block * i * 64,
				d_ctx, threads_per_block, d_extended);
		}
	}
	threads_per_block = total_datablocks - (i * total_threads);//remaining block
	//printf("The real threads_per_block is %d\n", threads_per_block);//total_datablocks
	sha1_kernel_global << <total_datablocks, proc >> >(d_message + total_threads * i * 64, 
		d_ctx, threads_per_block, d_extended);

	finish3 = clock();
	A1.kerneltime = (finish3 - start3) / CLOCKS_PER_SEC;
	//copy data form deivce to Host
	hipMemcpy(&ctx, d_ctx, sizeof(sha1_gpu_context), hipMemcpyDeviceToHost);
	//output the hash
	PUT_UINT32_BE(ctx.state[0], output, 0);
	PUT_UINT32_BE(ctx.state[1], output, 4);
	PUT_UINT32_BE(ctx.state[2], output, 8);
	PUT_UINT32_BE(ctx.state[3], output, 12);
	PUT_UINT32_BE(ctx.state[4], output, 16);
	hipFree(d_message);
	hipFree(d_ctx);
	hipFree(d_extended);
	printf("malloc process needs %f seconds,copy process needs %f seconds,kernel process needs %f seconds\n", A1.malloctime, A1.copytime, A1.kerneltime);
}