#include "hip/hip_runtime.h"
#include"time.h"
#include <string.h>
#include <stdio.h>
#include "Common.h"
#include"hip/hip_runtime.h"
#include""
#include <iostream>
#include <fstream>
#include<hip/hip_runtime.h>
#include<openssl\bn.h>
#include<openssl\sha.h>
#include"malloc.h"
#pragma comment(lib, "libeay32.lib")
#pragma comment(lib, "ssleay32.lib")
using namespace std;
class sha1gpu {
public:
	unsigned char block[64];
};
class Hash {
public:
	unsigned char hash[20];
};
#define SHA1CircularShift(bits,word) \
                ((((word) << (bits)) & 0xFFFFFFFF) | \
                ((word) >> (32-(bits))))

typedef struct {
	unsigned long total[2];     /* number of bytes processed  */
	unsigned long state[5];     /* intermediate digest state  */
	unsigned char buffer[64];   /* data block being processed */
} sha1_gpu1_context;


__constant__ static const unsigned char sha1_padding[64] =
{
	0x80, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
	0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
	0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
	0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0
};


/*
* Prepare SHA-1 for execution.
*/
__device__ void sha1_cpu_starts(sha1_gpu1_context* ctx)
{
	ctx->total[0] = 0;
	ctx->total[1] = 0;
	ctx->state[0] = 0x67452301;
	ctx->state[1] = 0xEFCDAB89;
	ctx->state[2] = 0x98BADCFE;
	ctx->state[3] = 0x10325476;
	ctx->state[4] = 0xC3D2E1F0;
}
__device__ unsigned long K[] =
{
	0x5A827999,
	0x6ED9EBA1,
	0x8F1BBCDC,
	0xCA62C1D6
};
__device__ static void sha1_cpu_upfoldprocess(sha1_gpu1_context *ctx, unsigned char data[64]) {
	
	int  t;                  /* Loop counter                 */
	unsigned long temp;               /* Temporary word value         */
	unsigned long W[80];              /* Word sequence                */
	unsigned long A, B, C, D, E;      /* Word buffers                 */
	A = ctx->state[0];
	B = ctx->state[1];
	C = ctx->state[2];
	D = ctx->state[3];
	E = ctx->state[4];
	unsigned long L1, L2;
	for (t = 1; t < 20; t = t + 2)
	{
	L1 = ((B&C)|((~B)&D))+E;
	L2 = ((A & SHA1CircularShift(30, B)) | ((~A) & C)) + D;
	E = C;
	D = SHA1CircularShift(30, B);
	C = SHA1CircularShift(30, A);
	if (t <= 15) {
		GET_UINT32_BE(W[t - 1], data, (t - 1) * 4);
		GET_UINT32_BE(W[t], data, t * 4);
	}
	else {
		W[t - 1] = SHA1CircularShift(1, W[t - 4] ^ W[t - 9] ^ W[t - 15] ^ W[t - 17]);
		W[t] = SHA1CircularShift(1, W[t-3] ^ W[t-8] ^ W[t-14] ^ W[t-16]);
	}
	B = SHA1CircularShift(5, A) + L1+ W[t - 1] + K[0];
	temp = SHA1CircularShift(5, B) + L2+ W[t] + K[0];
	temp &= 0xFFFFFFFF;
	A = temp;
	}
		
//(B ^ C ^ D) F2
	for (t = 21; t < 40; t = t + 2)
	{
	L1 = (B^C^D) + E;
	L2 = ((A^SHA1CircularShift(30, B)) ^ C) + D;
	E = C;
	D = SHA1CircularShift(30, B);
	C = SHA1CircularShift(30, A);
	W[t - 1] = SHA1CircularShift(1, W[t - 4] ^ W[t - 9] ^ W[t - 15] ^ W[t - 17]);
	W[t] = SHA1CircularShift(1, W[t - 3] ^ W[t - 8] ^ W[t - 14] ^ W[t - 16]);
	B = SHA1CircularShift(5, A) + L1+ W[t - 1] + K[1];
	temp = SHA1CircularShift(5, B) + L2 + W[t] + K[1];
	temp &= 0xFFFFFFFF;
	A = temp;
	}
		
//((B & C) | (B & D) | (C & D)) F3
	for (t = 41; t < 60; t = t + 2)
	{
	L1 = ((B & C) | (B & D) | (C & D)) + E;
	L2 = ((temp&SHA1CircularShift(30, B)) | (A&C) | (SHA1CircularShift(30, B)&C)) + D;
	E = C;
	D = SHA1CircularShift(30, B);
	C = SHA1CircularShift(30, A);//bcd
	W[t - 1] = SHA1CircularShift(1, W[t - 4] ^ W[t - 9] ^ W[t - 15] ^ W[t - 17]);
	W[t] = SHA1CircularShift(1, W[t - 3] ^ W[t - 8] ^ W[t - 14] ^ W[t - 16]);
	B = SHA1CircularShift(5, A) + L1 + W[t - 1] + K[2];
	temp = SHA1CircularShift(5, B) + L2+ W[t] + K[2];
	temp &= 0xFFFFFFFF;
	A = temp;
	}
		
//(B ^ C ^ D)
	for (t = 61; t < 80; t = t + 2)
	{
	L1 = (B^C^D) + E;
	L2 = ((A^SHA1CircularShift(30, B)) ^ C) + D;
	E = C;
	D = SHA1CircularShift(30, B);
	C = SHA1CircularShift(30, A);//bcd
	W[t - 1] = SHA1CircularShift(1, W[t - 4] ^ W[t - 9] ^ W[t - 15] ^ W[t - 17]);
	W[t] = SHA1CircularShift(1, W[t - 3] ^ W[t - 8] ^ W[t - 14] ^ W[t - 16]);
	B = SHA1CircularShift(5, A) + L1 + W[t - 1] + K[3];
	temp = SHA1CircularShift(5, B) + L2+ W[t] + K[3];
	temp &= 0xFFFFFFFF;
	A = temp;
	}
	//End
	ctx->state[0] += A;
	ctx->state[1] += B;
	ctx->state[2] += C;
	ctx->state[3] += D;
	ctx->state[4] += E;
}
/*
* Splits input message into blocks and processes them one by one. Also
* checks how many 0 need to be padded and processes the last, padded, block.
*/
__device__ void sha1_cpu_update(sha1_gpu1_context *ctx, unsigned char *input, int ilen)
{
	int fill;
	unsigned long left;
	if (ilen <= 0)
		return;
	left = ctx->total[0] & 0x3F;
	fill = 64 - left;
	ctx->total[0] += ilen;
	ctx->total[0] &= 0xFFFFFFFF;
	if (ctx->total[0] < (unsigned long)ilen)
		ctx->total[1]++;
	if (left && ilen >= fill) {
		memcpy((void *)(ctx->buffer + left), (void *)input, fill);
		sha1_cpu_upfoldprocess(ctx, ctx->buffer);
		input += fill;
		ilen -= fill;
		left = 0;
	}
	while (ilen >= 64) {
		sha1_cpu_upfoldprocess(ctx, input);
		input += 64;
		ilen -= 64;
	}
	if (ilen > 0) {
		memcpy((void *)(ctx->buffer + left), (void *)input, ilen);
	}
}


/*
* Process padded block and return hash to user.
*/
__device__ void sha1_cpu_finish(sha1_gpu1_context *ctx, unsigned char *output)
{
	unsigned long last, padn;
	unsigned long high, low;
	unsigned char msglen[8];


	high = (ctx->total[0] >> 29) | (ctx->total[1] << 3);
	low = (ctx->total[0] << 3);

	PUT_UINT32_BE(high, msglen, 0);
	PUT_UINT32_BE(low, msglen, 4);

	last = ctx->total[0] & 0x3F;
	padn = (last < 56) ? (56 - last) : (120 - last);

	sha1_cpu_update(ctx, (unsigned char *)sha1_padding, padn);
	sha1_cpu_update(ctx, msglen, 8);

	PUT_UINT32_BE(ctx->state[0], output, 0);
	PUT_UINT32_BE(ctx->state[1], output, 4);
	PUT_UINT32_BE(ctx->state[2], output, 8);
	PUT_UINT32_BE(ctx->state[3], output, 12);
	PUT_UINT32_BE(ctx->state[4], output, 16);
}

/*
* Execute SHA-1
*/
__device__ void sha1_cpu1(unsigned char *input, int ilen, unsigned char *output) {
	sha1_gpu1_context ctx;
	sha1_cpu_starts(&ctx);
	sha1_cpu_update(&ctx, input, ilen);
	sha1_cpu_finish(&ctx, output);
	memset(&ctx, 0, sizeof(sha1_gpu1_context));
}
__global__ void multisha1_thread(sha1gpu input[], int ilen, Hash output[], int n) {
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int i1 = blockDim.x*gridDim.x;
	for (int t = i; t < n; t = t + i1) {
		sha1_cpu1(input[t].block, ilen, output[t].hash);
	}
}
int main() {
	hipSetDevice(0);
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	int num_sm = prop.multiProcessorCount;
	printf("The num_sm of target is:%d\n", num_sm);
	dim3 ThreadperBlock(1024);
	dim3 BlockperGrid(num_sm);
	int length;
	cout << "INPUT THE SIZE" << endl;
	scanf("%d", &length);
	FILE*fp = fopen("1.txt", "w");
	BIGNUM*A = BN_new();
	for (int w = 0; w < 5; w++) {
		BN_rand(A, 256 * length, 1, 0);
		char*A1 = BN_bn2hex(A);
		fprintf(fp, "%s", A1);
	}
	cout << "Random file completed!" << endl;
	fclose(fp);
	int datablock = 5 * length;
	sha1gpu*sha1array;
	sha1array = new sha1gpu[datablock]; 
	printf("Please set the blocksize:\n");
	int blocksize; scanf("%d",&blocksize);
	char *input1=new char[blocksize+1];
	ifstream ifs;
	ifs.open("1.txt", ios::binary);
	if (!ifs) {
		cerr << "Error!" << endl;
		exit(1);
	}
	for (int i = 0; i < datablock; i++) {
		ifs.read(input1, blocksize); input1[blocksize] = '\0';
		for (int j = 0; j < blocksize; j++) {
			sha1array[i].block[j] = (unsigned char)input1[j];
		}
	}
	sha1gpu*INCUDA; Hash*hash1, *hash2;
	hash1 = new Hash[datablock];
	hipMalloc((void**)&INCUDA, datablock * sizeof(class sha1gpu));
	hipMemcpy(INCUDA, sha1array, datablock * sizeof(class sha1gpu), hipMemcpyHostToDevice);
	hipMalloc((void**)&hash2, datablock * sizeof(class Hash));
	hipMemcpy(hash2, hash1, sizeof(class Hash)*datablock, hipMemcpyHostToDevice);
	int allthread = datablock;
	hipEvent_t start1;
	hipEventCreate(&start1);
	hipEvent_t stop1;
	hipEventCreate(&stop1);
	hipEventRecord(start1, NULL);
	multisha1_thread << <BlockperGrid, ThreadperBlock >> >(INCUDA, blocksize, hash2, allthread);
	hipEventRecord(stop1, NULL);
	hipEventSynchronize(stop1);
	float msecTotal1 = 0.0f, total;
	hipEventElapsedTime(&msecTotal1, start1, stop1);
	total = msecTotal1 / 1000;
	cout << "GPU Runtime：" << total << "seconds" << endl;
	long r = 1 << 23; 
	FILE* fp11 = NULL;
	int nFileLen = 0;
	fp11 = fopen("1.txt", "rb");
	if (fp11 == NULL)
	{
		cout << "can't open file" << endl;
		return 0;
	}
	fseek(fp11, 0, SEEK_END);  
	nFileLen = ftell(fp11); 
	cout << "The Bytes of file is: " << nFileLen << endl;
	fclose(fp11);
	cout << "Throught：" << nFileLen *8/ total / r /blocksize<< " Gbps" << endl;
	hipMemcpy(hash1, hash2, sizeof(class Hash)*datablock, hipMemcpyDeviceToHost);
	FILE*fp1 = fopen("hash.txt", "w");
	for (int i = 0; i < datablock; i++) {
		for (int j = 0; j < 20; j++) {
			fprintf(fp1, "%02x", hash1[i].hash[j]);
		}
		fprintf(fp1, "\n");
	}
	unsigned char hashcpu[20];
	clock_t start, finsh;
	start = clock();
	for (int i = 0; i < datablock; i++) {
		SHA(sha1array[i].block, blocksize, hashcpu);
	}
	finsh = clock();
	float cputime = (float)(finsh - start) / 1000;
	cout << "OpenSSL execute " << cputime << "second" << endl;
	hipFree(hash2); 
	hipFree(INCUDA);
	BN_free(A);
	return 1;
}
