#include "hip/hip_runtime.h"
#include"Common1.h"
#include"hip/hip_runtime.h"
#include"time.h"
#include""
#include"stdio.h"
#include<string>
#include"hip/hip_runtime.h"
#include"malloc.h"
#include"sha.h"
#pragma comment(lib, "libeay32.lib")
#pragma comment(lib, "ssleay32.lib")
#define MAX_THREADS_PER_BLOCK 1

int main(void) {

	printf("You need to input your data,first!\n");
	char*a = (char*)malloc(5000*sizeof(char));
	scanf("%s", a);
	unsigned char hash[20];
	clock_t start1, finish1;
	float costtime;
	start1 = clock();
	sha1_cpu((unsigned char*)a, strlen(a), hash);
	finish1 = clock();
	costtime = (float)(finish1 - start1)/CLOCKS_PER_SEC;
	printf("CPU run time %f seconds\n", costtime);
	printf("CPU execute result:\n");
	for (int i = 0; i < 20; i++) printf("%02x ",hash[i]);
	printf("\n");
	hipEvent_t start, stop;
	float costtime4 = 0.0;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	sha1_gpu_global1((unsigned char*)a, strlen(a), hash, MAX_THREADS_PER_BLOCK);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&costtime4, start, stop);
	printf("GPU need time is %f seconds\n", costtime4 /1000);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	printf("GPU execute result:\n");
	for (int i = 0; i < 20; i++) printf("%02x ", hash[i]);
	printf("\n");
	clock_t start3, finish3;
	float costtime3;
	start3= clock();
	gpu_sha1((unsigned char*)a, strlen(a), hash);
	finish3 = clock();
	costtime3 = (float)(finish3 - start3) / CLOCKS_PER_SEC;
	printf("CPU run time %f seconds\n", costtime3);
	printf("GPU2 execute result:\n");
	for (int i = 0; i < 20; i++) printf("%02x ", hash[i]);
	printf("\n");
	clock_t start2, finish2;
	float costtime2;
	start2 = clock();//unsigned char*
	SHA1((unsigned char*)a, strlen(a), hash);
	//SHA1(b,64,hash);
	finish2 = clock();
	costtime2 = (float)(finish2 - start2) / CLOCKS_PER_SEC;
	printf("OpenSSL run time %f seconds\n", costtime2);
	printf("OpenSSL execute result:\n");
	for (int i = 0; i < 20; i++) printf("%02x ", hash[i]);
	printf("\n");
}
