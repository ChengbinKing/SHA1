#include "hip/hip_runtime.h"
#include"Common1.h"
#include"hip/hip_runtime.h"
#include"time.h"
#include""
#include"stdio.h"
#include<string>
#include"hip/hip_runtime.h"
#include"malloc.h"
#include<openssl\sha.h>
#include<openssl\bn.h>
#pragma comment(lib, "libeay32.lib")
#pragma comment(lib, "ssleay32.lib")
#define MAX_THREADS_PER_BLOCK 1
int main() {
	
	int n;
	printf("Please set the number of inputs\n");
	scanf("%d", &n);
	printf("Then you need to input the message\n");
	int*ilen1 = (int*)malloc(sizeof(int) * n);
	unsigned char**input = (unsigned char**)malloc(sizeof(unsigned char*) * n);
	unsigned char**hash = (unsigned char**)malloc(sizeof(unsigned char*)*n);
	char**INPUT = (char**)malloc(sizeof(char*)*n);
	for (int i = 0; i < n; i++) {
		input[i] = (unsigned char*)malloc(sizeof(unsigned char) * 512);
		hash[i] = (unsigned char*)malloc(sizeof(unsigned char) * 512);
		INPUT[i] = (char*)malloc(sizeof(char) * 1000);
	}
	BIGNUM*A = BN_new();
	printf("Random Data is preparing:\n");
	for (int i = 0; i < n; i++) {
		//scanf("%s", INPUT[i]);
		BN_rand(A, 512, 1, 0);
		INPUT[i]= BN_bn2hex(A);
		ilen1[i] = strlen(INPUT[i]);
		input[i] = (unsigned char*)INPUT[i];
		printf("%s\n", INPUT[i]);
	}
	BN_free(A);
	clock_t start1, finish1;
	float costtime;
	start1 = clock();
	for (int i = 0; i < n; i++) {
		SHA1(input[i], ilen1[i], hash[i]);
	}
	finish1 = clock();
	costtime = (float)(finish1 - start1);
	printf("OpenSSL run time %f ms\n", costtime);
	printf("OpenSSL results:\n");
	//multisha1_gpu(input, ilen1, hash,n);
	for (int i = 0; i < n; i++) {
		for (int j = 0; j < 20; j++) {
			printf("%02x", hash[i][j]);
		}
		printf("\n");
	}
	hipEvent_t start, stop;
	float costtime4 = 0.0;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	multisha1_gpu(input, ilen1, hash, n);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&costtime4, start, stop);
	printf("GPU need time is %f ms\n", costtime4);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	printf("GPU Result:\n");
	for (int i = 0; i < n; i++) {
		for (int j = 0; j < 20; j++) {
			printf("%02x", hash[i][j]);
		}
		printf("\n");
	}
	printf("OK\n");
	return 0;
}
