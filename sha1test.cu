#include "hip/hip_runtime.h"
#include"Common1.h"
#include"hip/hip_runtime.h"
#include"time.h"
#include""
#include"stdio.h"
#include<string>
#include"hip/hip_runtime.h"
#include"malloc.h"
#include"sha.h"
#pragma comment(lib, "libeay32.lib")
#pragma comment(lib, "ssleay32.lib")
#define MAX_THREADS_PER_BLOCK 1
void in1() {
	printf("You need to input your data,first!\n");
	char*a = (char*)malloc(5000*sizeof(char));
	scanf("%s", a);
	unsigned char hash[20];
	clock_t start1, finish1;
	float costtime;
	start1 = clock();
	sha1_cpu((unsigned char*)a, strlen(a), hash);
	finish1 = clock();
	costtime = (float)(finish1 - start1)/CLOCKS_PER_SEC;
	printf("CPU run time %f seconds\n", costtime);
	printf("CPU execute result:\n");
	for (int i = 0; i < 20; i++) printf("%02x ",hash[i]);
	printf("\n");
	hipEvent_t start, stop;
	float costtime4 = 0.0;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	sha1_gpu_global1((unsigned char*)a, strlen(a), hash, MAX_THREADS_PER_BLOCK);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&costtime4, start, stop);
	printf("GPU need time is %f seconds\n", costtime4 /1000);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	printf("GPU execute result:\n");
	for (int i = 0; i < 20; i++) printf("%02x ", hash[i]);
	printf("\n");
	/*clock_t start3, finish3;
	float costtime3;
	start3= clock();
	gpu_sha1((unsigned char*)a, strlen(a), hash);
	finish3 = clock();
	costtime3 = (float)(finish3 - start3) / CLOCKS_PER_SEC;
	printf("CPU run time %f seconds\n", costtime3);
	printf("GPU2 execute result:\n");
	for (int i = 0; i < 20; i++) printf("%02x ", hash[i]);
	printf("\n");*/
	clock_t start2, finish2;
	float costtime2;
	start2 = clock();//unsigned char*
	SHA1((unsigned char*)a, strlen(a), hash);
	//SHA1(b,64,hash);
	finish2 = clock();
	costtime2 = (float)(finish2 - start2) / CLOCKS_PER_SEC;
	printf("OpenSSL run time %f seconds\n", costtime2);
	printf("OpenSSL execute result:\n");
	for (int i = 0; i < 20; i++) printf("%02x ", hash[i]);
	printf("\n");
}
int main() {
	printf("You need to input your data,first!\n");
	int n = 5;
	int*ilen1 = (int*)malloc(sizeof(int) * n);
	unsigned char**input = (unsigned char**)malloc(sizeof(unsigned char*) * n);
	unsigned char**hash = (unsigned char**)malloc(sizeof(unsigned char*)*n);
	char**INPUT = (char**)malloc(sizeof(char*)*n);
	for (int i = 0; i < n; i++) {
		input[i] = (unsigned char*)malloc(sizeof(unsigned char) * 512);
		hash[i] = (unsigned char*)malloc(sizeof(unsigned char) * 512);
		INPUT[i] = (char*)malloc(sizeof(char) * 1000);
	}
	for (int i = 0; i < n; i++) {
		scanf("%s", INPUT[i]);
		ilen1[i] = strlen(INPUT[i]);
		input[i] = (unsigned char*)INPUT[i];
	}
	for (int i = 0; i < n; i++) {
		sha1_cpu(input[i], ilen1[i], hash[i]);
	}
	//multisha1_gpu(input, ilen1, hash,n);
	for (int i = 0; i < n; i++) {
		for (int j = 0; j < 20; j++) {
			printf("%02x", hash[i][j]);
		}
		printf("\n");
	}
	multisha1_gpu(input, ilen1, hash, n);

	for (int i = 0; i < n; i++) {
		for (int j = 0; j < 20; j++) {
			printf("%02x", hash[i][j]);
		}
		printf("\n");
	}
	printf("OK\n");
	return 0;
}
